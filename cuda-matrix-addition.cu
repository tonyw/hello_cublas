#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CPU 
void randomArray(float *cpu_arrayA,float *cpu_arrayB, unsigned long SQWIDTH) {
	srand((unsigned) time(NULL));
	for(unsigned long i = 0; i<SQWIDTH*SQWIDTH; ++i){
		cpu_arrayA[i] = ((float)rand()/(float)(RAND_MAX)) * 100;
		cpu_arrayB[i] = ((float)rand()/(float)(RAND_MAX)) * 100;
		//printf("Matrx [%d][%d]: %.2f\n", i,j, cpu_array[i][j]);
	}
}

void printResults(float *h_matA, float *h_matB, float *h_matC,int SQWIDTH){
	printf("Matrix A:\n");
	for(int i=0; i< SQWIDTH*SQWIDTH; i++){
		// int id = i + floor(i / (int)SQWIDTH )* (int)SQWIDTH;
		printf("%.2f	", h_matA[i]);
		if( (i+1) % SQWIDTH  == 0 ){
			printf("\n");
		}

	}
	printf("Matrix B:\n");
	for(int i=0; i< SQWIDTH*SQWIDTH; i++){
		// int id = i + floor(i / (int)SQWIDTH )* (int)SQWIDTH;
		printf("%.2f	", h_matB[i]);
		if( (i+1) % SQWIDTH  == 0 ){
			printf("\n");
		}

	}
	printf("Matrix C:\n");
	for(int i=0; i< SQWIDTH*SQWIDTH; i++){
		// int id = i + floor(i / (int)SQWIDTH )* (int)SQWIDTH;
		printf("%.2f	", h_matC[i]);
		if( (i+1) % SQWIDTH  == 0 ){
			printf("\n");
		}

	}
}

// GPU
__global__ void kernel_1t1e(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: Device a row major indexing
	int rowID = threadIdx.y + blockIdx.y * blockDim.y; 	// Row address
	int colID = threadIdx.x + blockIdx.x * blockDim.x;	// Column Address
	int elemID;											// Element address

    // a_ij = a[i][j], where a is in row major order
	if(rowID < WIDTH && colID < WIDTH){
		elemID = colID + rowID * WIDTH; 				
		C[elemID] = A[elemID] + B[elemID];
	}
}

__global__ void kernel_1t1r(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: Each thread = 1 output row
	int rowID = threadIdx.y + blockIdx.y * blockDim.y;	// Row address

	if(rowID < WIDTH) {
		for(int i = 0; i<WIDTH; i++){
			//elemID = colID + rowID * WIDTH; 
			C[i + rowID*WIDTH] = A[i + rowID*WIDTH] + B[i + rowID*WIDTH];
		}
	}
}

__global__ void kernel_1t1c(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: Each thread = 1 output row
	int colID = threadIdx.x + blockIdx.x * blockDim.x;	// Row address

	if(colID < WIDTH) {
		for(int i = 0; i<WIDTH; i++){
			//elemID = colID + rowID * WIDTH; 
			C[colID + i*WIDTH] = A[colID + i*WIDTH] + B[colID + i*WIDTH];
		}
	}
}


int main(int argv, int* argc[]) {
	// Memory specification
	unsigned long SQWIDTH;
	scanf("%lu", &SQWIDTH);

	const size_t d_size = sizeof(float) * size_t(SQWIDTH*SQWIDTH);

	// Multiprocessing constants
	const dim3 threadsPerBlock(32,32); 	// Must not exceed 1024 (max thread per block)
	const dim3 blocksPerGrid(ceil(SQWIDTH/32),ceil(SQWIDTH/32));		// Number of blocks that will be used

	// CUDA TIME
	float ms;
	float avems = 0.0;
	hipEvent_t start,end;



	// Initialize host matrices
	clock_t h_alloctime = clock();
	float *h_matA = (float*) malloc(SQWIDTH*SQWIDTH * sizeof(float));
	float *h_matB = (float*) malloc(SQWIDTH*SQWIDTH * sizeof(float));
	float *h_matC = (float*) malloc(SQWIDTH*SQWIDTH * sizeof(float));
	randomArray(h_matA, h_matB, SQWIDTH);
	printf("[**] CPU Allocation time for %dx%d matrix: %.6fsec \n",SQWIDTH,SQWIDTH,(double)(clock()-h_alloctime)/CLOCKS_PER_SEC );
	// Initialize device matrices
	float *d_matA, *d_matB, *d_matC;
	
	clock_t d_alloctime = clock();
	hipMalloc((void **) &d_matA, d_size);
	hipMalloc((void **) &d_matB, d_size);
	hipMalloc((void **) &d_matC, d_size);
	hipMemcpy(d_matA, h_matA, d_size, hipMemcpyHostToDevice);
	hipMemcpy(d_matB, h_matB, d_size, hipMemcpyHostToDevice); 
	printf("[**] GPU Allocation time for %lux%lu matrix: %.6fsec \n",SQWIDTH,SQWIDTH,(double)(clock()-d_alloctime)/CLOCKS_PER_SEC );


	// Number of threads = SQWIDTH*SQWIDTH
	printf("[**] Starting kernel program 'kernel_1t1e' execution\n");
	for(int i = 0; i<10; i++){
		// ELEMENT
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		kernel_1t1e<<< blocksPerGrid, threadsPerBlock >>>(d_matA,d_matB,d_matC,SQWIDTH);


		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		printf("\tIteration no. %d: %.6fsecs\n", i, ms);
		avems+=ms;
		hipMemcpy(h_matC, d_matC, d_size, hipMemcpyDeviceToHost); 

		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("[**] Average kernel execution time: %.2fsec.\n\n", avems/10.0);
	

	printf("[] Starting kernel program 'kernel_1t1r'.execution\n");
	avems = 0.0;
	for(int i = 0; i<10; i++){
		// ELEMENT
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		kernel_1t1r<<< blocksPerGrid, threadsPerBlock >>>(d_matA,d_matB,d_matC,SQWIDTH);


		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		printf("\tIteration no. %d: %.6fsecs\n", i, ms);
		avems+=ms;
		hipMemcpy(h_matC, d_matC, d_size, hipMemcpyDeviceToHost); 
		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("[**] Average kernel execution time: %.2fsec.\n\n", avems/10.0);
	
	printf("[**] Starting kernel program 'kernel_1t1c' execution\n");
	avems = 0;
	for(int i = 0; i<10; i++){
		// ELEMENT
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		kernel_1t1c<<< blocksPerGrid, threadsPerBlock >>>(d_matA,d_matB,d_matC,SQWIDTH);


		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		printf("\tIteration no. %d: %.6fsecs\n", i, ms);
		avems+=ms;
		hipMemcpy(h_matC, d_matC, d_size, hipMemcpyDeviceToHost); 

		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("[**] Average kernel execution time: %.2fsec.\n", avems/10.0);
	
	hipFree(d_matA);
	hipFree(d_matB);
	hipFree(d_matC);
	free(h_matA);
	free(h_matB);
	free(h_matC);

	return 0;
}