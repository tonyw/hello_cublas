#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_fp16.h>

inline hipError_t cuda_check(hipError_t err){
    if(err != hipSuccess){
        printf("cuda error: %s\n", hipGetErrorString(err));
        assert(false);
    }
    return err;
}

void cpu_init(half *a, half *b, half *c_, size_t N){
    for(size_t c=0;c<N;c++){
        for(size_t r=0;r<N;r++){
            a[c*N+r] = __float2half((float)r);
            b[c*N+r] = __float2half((float)c);
            c_[c*N+r] = __float2half(0.0f);
            //printf("[%d,%d]: a:%d, b:%d\n", r,c,(int)__half2float(a[r*N+c]), (int)__half2float(b[r*N+c]));
        }
    }
}

// 将CPU上的矩阵A和B复制到GPU内存
void gpu_init(half *cpu_a, half *cpu_b, half *gpu_a, half *gpu_b, size_t N){
    cuda_check(hipMemcpy(gpu_a, cpu_a, N*N*sizeof(half), hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(gpu_b, cpu_b, N*N*sizeof(half), hipMemcpyHostToDevice));
}

__global__ void check_matrix_multiply_1t1e(half *a, half *b, half *c_gpu, half *epsilon,int *flag, size_t N){
    
    /*
    tidx.[y,x]: 0,2, cidx.[r,c]: [0, 2],i: 0,a_idx:0,a_data: 0, b_idx:8,b_data: 2
    tidx.[y,x]: 0,2, cidx.[r,c]: [0, 2],i: 1,a_idx:4,a_data: 0, b_idx:9,b_data: 2
    tidx.[y,x]: 0,2, cidx.[r,c]: [0, 2],i: 2,a_idx:8,a_data: 0, b_idx:10,b_data: 2
    tidx.[y,x]: 0,2, cidx.[r,c]: [0, 2],i: 3,a_idx:12,a_data: 0, b_idx:11,b_data: 2
     */
    int r_index = blockDim.y * blockIdx.y + threadIdx.y;
    int c_index = blockDim.x * blockIdx.x + threadIdx.x;
    half temp = 0;
    if(r_index < N && c_index < N){
        for(int i = 0; i < N; i++){
            int a_idx = i*(int)N+r_index;
            int b_idx = c_index*(int)N+i;
            half a_data = a[a_idx];
            half b_data = b[b_idx];
            temp = __hadd(temp, __hmul(a_data, b_data));
            if(r_index == 15 && c_index == 11){
                printf("tidx.[y,x]: %d,%d, cidx.[r,c]: [%d, %d],i: %d,a_idx:%d,a_data: %d, b_idx:%d,b_data: %d, temp: %f.05\n",threadIdx.y, threadIdx.x, r_index, c_index,i, a_idx, (int)__half2float(a[a_idx]), b_idx, (int)__half2float(b_data), __half2float(temp));
            }
            
        }
        if(r_index == 15 && c_index == 11){
            printf("temp: %f.05, gpu_c: %f.05\n", __half2float(temp), __half2float(c_gpu[r_index*N+c_index]));
        }
        if(__habs(temp - c_gpu[r_index*N+c_index]) > *epsilon){
            flag[r_index*N+c_index] = 1;
        }
    }
}


bool check_gpu_multiply(half *a, half *b, half *c_gpu, int N){
    const size_t threads_per_block = 32;
    const dim3 threads(threads_per_block,threads_per_block);
    //printf("CPU: Before kernel launch\n");
    //printf("threads: %d, %d\n", threads.x, threads.y);
    const dim3 blocks((N+threads.x-1)/threads.x,(N+threads.y-1)/threads.y);   
    //printf("blocks: %d, %d\n", blocks.x, blocks.y);
    
    half* EPSILON;
    cuda_check(hipMallocManaged((void**)&EPSILON, sizeof(half)));
    *EPSILON = __float2half(1e-5f);
    int *flag;
    cuda_check(hipMalloc(&flag, N*N*sizeof(int)));
    cuda_check(hipMemset(flag, 0, N*N*sizeof(int)));
    //printf("CPU: Launching kernel...\n");
    
    check_matrix_multiply_1t1e<<<blocks,threads>>>(a, b, c_gpu, EPSILON, flag, N);
    
    cuda_check(hipGetLastError());
    cuda_check(hipDeviceSynchronize());
    //printf("CPU: After kernel execution\n");
    
    int *flag_host = (int*)malloc(N*N*sizeof(int));
    cuda_check(hipMemcpy(flag_host, flag, N*N*sizeof(int), hipMemcpyDeviceToHost));
    for(int i = 0; i < N*N; i++){
        if(flag_host[i] == 1){
            printf("gpu error: %d\n", i);
            cuda_check(hipFree(flag));
            free(flag_host);
            return false;
        }
    }
    cuda_check(hipFree(flag));
    cuda_check(hipFree(EPSILON));
    free(flag_host);
    return true;
}





// Helper function to convert cuBLAS status to a readable string
const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "CUBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "CUBLAS_STATUS_LICENSE_ERROR";
        default:
            return "UNKNOWN CUBLAS STATUS";
    }
}


// Function to perform matrix multiplication using cuBLAS
void matrix_multiply_cublas(half *a, half *b, half *c_gpu, size_t N) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);

    hipblasStatus_t status = hipblasHgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        N, N, N,
        &alpha,
        a, N,
        b, N,
        &beta,
        c_gpu, N
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS matrix multiplication failed\n");
        fprintf(stderr, "code: %d, status: %s\n", status, cublasGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    hipblasDestroy(handle);
}

// 在CPU上分配页锁定内存(pinned memory)
void allocate_memory_cpu(half **a, half **b, half **c, size_t size) {
    cuda_check(hipHostMalloc(a, size, hipHostMallocDefault));
    cuda_check(hipHostMalloc(b, size, hipHostMallocDefault));
    cuda_check(hipHostMalloc(c, size, hipHostMallocDefault));
    if (*a == NULL || *b == NULL || *c == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
}

// 在GPU上分配设备内存
void allocate_memory_gpu(half **a, half **b, half **c, size_t size) {
    cuda_check(hipMalloc(a, size));
    cuda_check(hipMalloc(b, size));
    cuda_check(hipMalloc(c, size));
    if (*a == NULL || *b == NULL || *c == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
}

// 主函数：
// 1. 打印GPU设备信息
// 2. 分配内存并初始化数据
// 3. 执行GPU矩阵加法并计时
// 4. 验证结果并清理内存
int main(){
    int gpu_index=0;
    hipGetDevice(&gpu_index);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, gpu_index);
    printf("GPU name: %s\n", prop.name);
    printf("GPU compute capability: %d.%d\n", prop.major, prop.minor);
    printf("GPU sm count: %d\n", prop.multiProcessorCount);
    printf("GPU global memory: %zu GB\n", prop.totalGlobalMem/1024/1024/1024);
    printf("GPU shared memory per block: %zu KB\n", prop.sharedMemPerBlock/1024);
    printf("GPU L2 cache size: %d KB\n", prop.l2CacheSize/1024);
    printf("GPU warp size: %d\n", prop.warpSize);
    printf("GPU maximum threads per block: %d\n", prop.maxThreadsPerBlock);

    const size_t N = 16;
    half *cpu_a,*cpu_b,*cpu_c;
    half *gpu_a,*gpu_b,*gpu_c;
    size_t size = N * N * sizeof(half);
    allocate_memory_gpu(&gpu_a, &gpu_b, &gpu_c, size);
    allocate_memory_cpu(&cpu_a, &cpu_b, &cpu_c, size);
    cpu_init(cpu_a,cpu_b,cpu_c, N);
    gpu_init(cpu_a,cpu_b, gpu_a, gpu_b, N);
    

    // CUDA TIME
    float ms;
    float avems = 0.0;
    hipEvent_t start,end;

    //warm up
    for(int i = 0;i < 3;i++){
        matrix_multiply_cublas(gpu_a, gpu_b, gpu_c, N);
        cuda_check(hipDeviceSynchronize());
    }
    
    for(int i = 0; i < 10; i++){

        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);

        matrix_multiply_cublas(gpu_a, gpu_b, gpu_c, N);
        //check_gpu_multiply(gpu_a, gpu_b, gpu_c, N);
        hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

        printf("\tIteration no. %d: %.2f ms\n", i, ms);
        avems+=ms;

        hipEventDestroy(start);
        hipEventDestroy(end);
    }
    printf("[**] Average kernel execution time: %.2fms.\n\n", avems/10.0);
    cuda_check(hipGetLastError());
    cuda_check(hipDeviceSynchronize());
    check_gpu_multiply(gpu_a, gpu_b, gpu_c, N)? printf("gpu ok\n") : printf("gpu error\n");
    cuda_check(hipFree(gpu_a));
    cuda_check(hipFree(gpu_b));
    cuda_check(hipFree(gpu_c));
    cuda_check(hipHostFree(cpu_a));
    cuda_check(hipHostFree(cpu_b));
    cuda_check(hipHostFree(cpu_c));
    return 0;
}


