#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_fp16.h>

// CUDA错误检查的辅助函数
inline hipError_t cuda_check(hipError_t err){
    if(err != hipSuccess){
        printf("cuda error: %s\n", hipGetErrorString(err));
        assert(false);
    }
    return err;
}

// 在CPU上初始化矩阵A、B和C
// a和b分别用行号和列号填充，c初始化为0
void cpu_init(half *a, half *b, half *c_, size_t N){
    for(size_t c=0;c<N;c++){
        for(size_t r=0;r<N;r++){
            a[r*N+c] = __float2half((float)r);
            b[r*N+c] = __float2half((float)c);
            c_[r*N+c] = __float2half(0.0f);
            //printf("cpu: %d, %d, %d\n", (int)__half2float(a[r*N+c]), (int)__half2float(b[r*N+c]), (int)__half2float(c_[r*N+c]));
        }
    }
}

// 将CPU上的矩阵A和B复制到GPU内存
void gpu_init(half *cpu_a, half *cpu_b, half *gpu_a, half *gpu_b, size_t N){
    cuda_check(hipMemcpy(gpu_a, cpu_a, N*N*sizeof(half), hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(gpu_b, cpu_b, N*N*sizeof(half), hipMemcpyHostToDevice));
}

__global__ void check_matrix_multiply_1t1e(half *a, half *b, half *c_gpu, half *epsilon,int *flag, size_t N){
    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        printf("GPU check_matrix_multiply_1t1e\n");
    }
    
    int c_index = blockDim.y * blockIdx.y + threadIdx.y;
    int r_index = blockDim.x * blockIdx.x + threadIdx.x;
    half temp = 0;
    if(r_index < N && c_index < N){
        temp = 0;
        for(int i = 0; i < N; i++){
            temp += (a[r_index*N+i] * b[i*N+c_index]);
            
        }
        printf("temp: %d, c_gpu[%d]: %d, epsilon: %f.2\n", (int)__half2float(temp), r_index*N+c_index,(int)__half2float(c_gpu[r_index*N+c_index]),__half2float(*epsilon));
        if(__habs(temp - c_gpu[r_index*N+c_index]) > *epsilon){
            flag[r_index*N+c_index] = 1;
        }
    }
}

bool check_gpu_multiply(half *a, half *b, half *c_gpu, int N){
    const size_t threads_per_block = 32;
    const dim3 threads(threads_per_block,threads_per_block);
    printf("CPU: Before kernel launch\n");
    printf("threads: %d, %d\n", threads.x, threads.y);
    const dim3 blocks((N+threads.x-1)/threads.x,(N+threads.y-1)/threads.y);   
    printf("blocks: %d, %d\n", blocks.x, blocks.y);
    
    half EPSILON = __float2half(1e-5f);
    int *flag;
    cuda_check(hipMalloc(&flag, N*N*sizeof(int)));
    cuda_check(hipMemset(flag, 0, N*N*sizeof(int)));
    printf("CPU: Launching kernel...\n");
    
    check_matrix_multiply_1t1e<<<blocks,threads>>>(a, b, c_gpu, &EPSILON, flag, N);
    
    cuda_check(hipGetLastError());
    cuda_check(hipDeviceSynchronize());
    printf("CPU: After kernel execution\n");
    
    int *flag_host = (int*)malloc(N*N*sizeof(int));
    cuda_check(hipMemcpy(flag_host, flag, N*N*sizeof(int), hipMemcpyDeviceToHost));
    for(int i = 0; i < N*N; i++){
        if(flag_host[i] == 1){
            printf("gpu error: %d\n", i);
            cuda_check(hipFree(flag));
            free(flag_host);
            return false;
        }
    }
    cuda_check(hipFree(flag));
    free(flag_host);
    return true;
}



// Helper function to convert cuBLAS status to a readable string
const char* cublasGetErrorString(hipblasStatus_t status) {
    switch (status) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "CUBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "CUBLAS_STATUS_LICENSE_ERROR";
        default:
            return "UNKNOWN CUBLAS STATUS";
    }
}


// Function to perform matrix multiplication using cuBLAS
void matrix_multiply_cublas(half *a, half *b, half *c_gpu, size_t N) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);

    hipblasStatus_t status = hipblasHgemm(
        handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_T,
        N, N, N,
        &alpha,
        a, N,
        b, N,
        &beta,
        c_gpu, N
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS matrix multiplication failed\n");
        fprintf(stderr, "code: %d, status: %s\n", status, cublasGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    hipblasDestroy(handle);
}

// 在CPU上分配页锁定内存(pinned memory)
void allocate_memory_cpu(half **a, half **b, half **c, size_t size) {
    cuda_check(hipHostMalloc(a, size, hipHostMallocDefault));
    cuda_check(hipHostMalloc(b, size, hipHostMallocDefault));
    cuda_check(hipHostMalloc(c, size, hipHostMallocDefault));
    if (*a == NULL || *b == NULL || *c == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
}

// 在GPU上分配设备内存
void allocate_memory_gpu(half **a, half **b, half **c, size_t size) {
    cuda_check(hipMalloc(a, size));
    cuda_check(hipMalloc(b, size));
    cuda_check(hipMalloc(c, size));
    if (*a == NULL || *b == NULL || *c == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
}

// 主函数：
// 1. 打印GPU设备信息
// 2. 分配内存并初始化数据
// 3. 执行GPU矩阵加法并计时
// 4. 验证结果并清理内存
int main(){
    int gpu_index=0;
    hipGetDevice(&gpu_index);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, gpu_index);
    printf("GPU name: %s\n", prop.name);
    printf("GPU compute capability: %d.%d\n", prop.major, prop.minor);
    printf("GPU sm count: %d\n", prop.multiProcessorCount);
    printf("GPU global memory: %zu GB\n", prop.totalGlobalMem/1024/1024/1024);
    printf("GPU shared memory per block: %zu KB\n", prop.sharedMemPerBlock/1024);
    printf("GPU L2 cache size: %d KB\n", prop.l2CacheSize/1024);
    printf("GPU warp size: %d\n", prop.warpSize);
    printf("GPU maximum threads per block: %d\n", prop.maxThreadsPerBlock);

    const size_t N = 4;
    half *cpu_a,*cpu_b,*cpu_c;
    half *gpu_a,*gpu_b,*gpu_c;
    size_t size = N * N * sizeof(half);
    allocate_memory_gpu(&gpu_a, &gpu_b, &gpu_c, size);
    allocate_memory_cpu(&cpu_a, &cpu_b, &cpu_c, size);
    cpu_init(cpu_a,cpu_b,cpu_c, N);
    gpu_init(cpu_a,cpu_b, gpu_a, gpu_b, N);
    

    // CUDA TIME
    float ms;
    float avems = 0.0;
    hipEvent_t start,end;

    //warm up
    for(int i = 0;i < 3;i++){
        matrix_multiply_cublas(gpu_a, gpu_b, gpu_c, N);
    }
    
    for(int i = 0; i < 10; i++){

        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, 0);

        matrix_multiply_cublas(gpu_a, gpu_b, gpu_c, N);
        hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

        printf("\tIteration no. %d: %.2f ms\n", i, ms);
        avems+=ms;

        hipEventDestroy(start);
        hipEventDestroy(end);
    }
    printf("[**] Average kernel execution time: %.2fms.\n\n", avems/10.0);
    cuda_check(hipGetLastError());
    cuda_check(hipDeviceSynchronize());
    check_gpu_multiply(gpu_a, gpu_b, gpu_c, N)? printf("gpu ok\n") : printf("gpu error\n");
    cuda_check(hipFree(gpu_a));
    cuda_check(hipFree(gpu_b));
    cuda_check(hipFree(gpu_c));
    cuda_check(hipHostFree(cpu_a));
    cuda_check(hipHostFree(cpu_b));
    cuda_check(hipHostFree(cpu_c));
    return 0;
}


